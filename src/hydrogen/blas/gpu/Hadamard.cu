#include "hip/hip_runtime.h"
#include <hydrogen/blas/gpu/Hadamard.hpp>

#include <El/hydrogen_config.h>
#ifdef HYDROGEN_HAVE_CUDA
#include <hydrogen/device/gpu/CUDA.hpp>
#include <hip/hip_runtime.h>
using gpuStream_t = hipStream_t;
#elif defined(HYDROGEN_HAVE_ROCM)
#include <hydrogen/device/gpu/ROCm.hpp>
#include <hip/hip_runtime.h>
using gpuStream_t = hipStream_t;
#endif

namespace
{

template <typename T>
__global__ void Hadamard1D_kernel(size_t size,
                                  T const* __restrict__ X,
                                  T const* __restrict__ Y,
                                  T* __restrict__ Z)
{
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t numThreads = blockDim.x * gridDim.x;
    for (size_t pos = tid; pos < size; pos += numThreads)
    {
        Z[pos] = X[pos] * Y[pos];
    }
}

template <typename T>
__global__ void MultAssign_kernel(size_t size, T const* X, T* Y)
{
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t numThreads = blockDim.x * gridDim.x;
    for (size_t pos = tid; pos < size; pos += numThreads)
    {
        Y[pos] *= X[pos];
    }
}

template <typename T>
__global__ void Hadamard2D_kernel(size_t height, size_t width,
                                  T const* X, size_t colStrideX, size_t rowStrideX,
                                  T const* Y, size_t colStrideY, size_t rowStrideY,
                                  T* Z, size_t colStrideZ, size_t rowStrideZ)
{
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t numThreads = blockDim.x * gridDim.x;
    for (size_t pos = tid; pos < height * width; pos += numThreads)
    {
        const size_t i = pos % height;
        const size_t j = pos / height;
        const auto& x_local = X[i*colStrideX+j*rowStrideX];
        const auto& y_local = Y[i*colStrideY+j*rowStrideY];
        Z[i*colStrideZ+j*rowStrideZ] = x_local * y_local;
    }
}

}// namespace <anon>

namespace hydrogen
{

template <typename T, typename>
void Hadamard_GPU_impl(
    size_t height, size_t width,
    T const* X, size_t colStrideX, size_t rowStrideX,
    T const* Y, size_t colStrideY, size_t rowStrideY,
    T* Z, size_t colStrideZ, size_t rowStrideZ,
    gpuStream_t stream)
{
    if (height <= 0 || width <= 0) { return; }
    size_t size = height * width;
    size_t const blockDim = 256;
    size_t const gridDim = (size + blockDim - 1) / blockDim;
    if (colStrideX == 1 && rowStrideX == height
        && colStrideY == 1 && rowStrideY == height
        && colStrideZ == 1 && rowStrideZ == height)
    {
        if (X == Z)
        {
            gpu::LaunchKernel(
                MultAssign_kernel<T>,
                gridDim, blockDim, 0, SyncInfo<Device::GPU>(stream, nullptr),
                size, Y, Z);
        }
        else if (Y == Z)
        {
            gpu::LaunchKernel(
                MultAssign_kernel<T>,
                gridDim, blockDim, 0, SyncInfo<Device::GPU>(stream, nullptr),
                size, X, Z);
        }
        else
        {
            gpu::LaunchKernel(
                Hadamard1D_kernel<T>,
                gridDim, blockDim, 0, SyncInfo<Device::GPU>(stream, nullptr),
                size, X, Y, Z);
        }
    }
    else
    {
        gpu::LaunchKernel(
            Hadamard2D_kernel<T>,
            gridDim, blockDim, 0, SyncInfo<Device::GPU>(stream, nullptr),
            height, width,
            X, colStrideX, rowStrideX,
            Y, colStrideY, rowStrideY,
            Z, colStrideZ, rowStrideZ);
    }

}

#define ETI(T)                                  \
    template void Hadamard_GPU_impl(            \
        size_t, size_t,                         \
        T const*, size_t, size_t,               \
        T const*, size_t, size_t,               \
        T*, size_t, size_t, gpuStream_t)

#ifdef HYDROGEN_GPU_USE_FP16
ETI(gpu_half_type);
#endif

ETI(float);
ETI(double);

}// namespace hydrogen
